#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <random>
#include <typeinfo>
#include <stdlib.h> 



using namespace std;

vector<vector<float>> load_data(26880);
vector<vector<float>> X_train(26880);
vector<vector<float>> x_val(8400);
vector<vector<float>> x_test(6720);
vector<float> y_train;
vector<float> y_val;
vector<float> y_test;

vector<float> losses, val_losses;
vector<vector<float>> w1(784), w2(10);
vector<float> b1(10), b2(10);
vector<vector<float>> output(10), a1_train(1, vector<float>(10, 0)), a1_test(1, vector<float>(10, 0)), a1_val(1, vector<float>(10, 0));
vector<float> b1_grad(10);
float b2_grad;
vector<vector<float>> w1_grad(10), w2_grad(10); // w2_grad (10, 10)

float* dev_w1; float* dev_out;float* dev_xx; 
float* dev_a1;float* dev_w2; float* dev_out2;
int xSize = 1*784 * sizeof(float);
int w1Size = 784*10 * sizeof(float);
int outSize = 1*10 * sizeof(float);

float* xcp; float* w1cp; float* outcp;
float* a1cp; float* w2cp; float* out2cp;


float l1 = 0, l2 = 0;



vector<vector<float>> transpose(vector<vector<float>>& a)
{
    vector<vector<float>> ret(a[0].size(), vector<float>(a.size()));
    for(int i=0;i<a.size();i++)
    {
        for(int j=0;j<a[i].size();j++)
        {
            ret[j][i] = a[i][j];
        }
    }
    return ret;
}
vector<vector<float>> dot(vector<vector<float>> &a, vector<vector<float>>& b)
{
    vector<vector<float>> out(a.size(), vector<float>(b[0].size(), 0.0f));

    for(int i=0;i<a.size();i++)
    {
        for(int j=0;j<b[0].size();j++)
        {
            for(int k=0;k<b.size();k++)
            {
                out[i][j] += a[i][k] * b[k][j];
            }
        }
    }
    return out;
}

__global__ void Dot2(float* a, float* b, float* out, int a_width, int b_width)
{
    int tid, tx, ty;
    tx = blockDim.x * blockIdx.x + threadIdx.x;
    ty = blockDim.y * blockIdx.y + threadIdx.y;
    int DimX = gridDim.x * blockDim.x;
    tid = DimX * ty + tx;

    float Value = 0; float AVal = 0; float BVal = 0;

    for (int k = 0; k < a_width; k++)
    {
        AVal = a[ty * a_width + k];
        BVal = b[k * b_width + tx];
        Value += AVal * BVal;
        // printf("\n tid: %d, k: %d, Value: %f, MVal: %f, NVal: %f ", tid, k, Value, AVal, BVal);
    }

    out[tid] = Value;
}


void summ(vector<vector<float>> a, vector<vector<float>> b, vector<vector<float>>& out){   
    for(int i=0; i<a.size(); i++)
    {
        for(int j=0; j<a[i].size(); j++)
        {
            out[i][j] = a[i][j] + b[i][j];
        }
    }
}


void sumb(vector<vector<float>> a, vector<float> b, vector<vector<float>>& out)
{
    for(int i=0;i<a.size();i++)
    {
        for(int j=0;j<a[i].size();j++)
        {
            out[i][j] = a[i][j] + b[j];
        }
    }
}


vector<vector<float>> sigmoid(vector<vector<float>> a){
    vector<vector<float>> out(a.size(), vector<float>(a[0].size(), 0));
    for(int i=0; i<a.size(); i++)
    {
        for(int j=0; j<a[i].size(); j++)
        {
            float tmp = a[i][j] / 4000;
            out[i][j] = 1.0/((float)1.0+exp(-tmp));
        }
    }
    return out;
}

void printResult(float* M, float* N, float* P)
{
    int row = 0; int col = 0;
    for (row = 0; row < 1; row++)
    {
        for (col = 0; col < 10; col++)
        {
            int Destindex = row * 1 + col;
            printf( "%f (= P[%d][%d]) = %f (= M[%d][%d]) + %f (= N[%d][%d]) \n", \
            P[Destindex], row, col, M[Destindex], row, col, N[Destindex], row, col );
        }
    }
}


vector<vector<float>> forward(vector<vector<float>> &x, vector<vector<float>> &w1, vector<vector<float>> &w2, vector<vector<float>> &a1){
    cout << "forward!" << endl;
    vector<vector<float>> out;

    xcp = (float*)malloc(xSize); //(1, 784)
    w1cp = (float*)malloc(w1Size); //(784, 10)
    outcp = (float*)malloc(outSize); //(1, 10)


    int xxsize = x[0].size();
    int wwsize = w1[0].size();
    for(int i=0; i<x.size(); i++)
    {
        for(int j=0; j<x[i].size(); j++)
        {
            xcp[i*xxsize + j] = x[i][j];
        }
    }
    for(int i=0; i<w1.size(); i++)
    {
        for(int j=0; j<w1[i].size(); j++)
        {

            w1cp[i*wwsize + j] = w1[i][j];
        }
    }

    for(int i=0; i<10; i++)
    {
        outcp[i] = 0;
    }

    // for(int i=0; i<10; i++)
    // {
    //     for(int j=0; j<10; j++)
    //     {
    //         cout << "값 복사: " << w1cp[i*wwsize + j]<<"=" << w1[i][j];
    //     }
    // }

    hipMalloc((void**)&dev_xx, xSize);
    hipMalloc((void**)&dev_w1, w1Size);
    hipMalloc((void**)&dev_out, outSize);

    hipMemcpy(dev_xx, xcp, xSize, hipMemcpyHostToDevice);
    hipMemcpy(dev_w1, w1cp, w1Size, hipMemcpyHostToDevice);
    hipMemcpy(dev_out, outcp, outSize, hipMemcpyHostToDevice);

    dim3 Dg(1, 10, 1);
    dim3 Db(128, 1, 1);
    cout << "첫번째 dot" <<endl;
    Dot2 <<<Dg, Db>>> (dev_xx, dev_w1, dev_out, 784, 10);
    hipDeviceSynchronize();
    hipMemcpy(outcp, dev_out, outSize, hipMemcpyDeviceToHost);

    out.resize(1);
    for(int i=0; i<10; i++)
    {
        out[0].push_back((float)outcp[i]);
        //cout << "out: " << out[0][i] << " " << typeid(out[0][i]).name() << endl;
    }

    // printResult(xcp, w1cp, outcp);

    hipFree(dev_xx); hipFree(dev_w1); hipFree(dev_out);
    free(xcp); free(w1cp); free(outcp);

    cout << out.size() << " " << out[0].size() << " " << outSize/sizeof(float) << endl;

    sumb(out, b1, out); //(1, 10)

    cout << out.size() << " " << out[0].size() << " " << outSize/sizeof(float) << endl;

    //cout << out[0][0] << endl;
    a1 = sigmoid(out); //(1,10)

    cout << out.size() << " " << out[0].size() << " " << outSize/sizeof(float) << endl;

    //-----------------------------------------------------------------------------------------------------------------------------두번째 dot
    int a1cp_Size = 10*sizeof(float);
    int w2cp_Size = 100*sizeof(float);

    a1cp = (float*)malloc(a1cp_Size); //(1,10)
    w2cp = (float*)malloc(w2cp_Size); //(10, 10)
    out2cp = (float*)malloc(outSize); //(1, 10)

    int a1size = a1[0].size();
    int w2size = w2[0].size();

    for(int i=0; i<a1.size(); i++)
    {
        for(int j=0; j<a1[i].size(); j++)
        {
            a1cp[i*a1size + j] = a1[i][j];
        }
    }
    for(int i=0; i<w2.size(); i++)
    {
        for(int j=0; j<w2[i].size(); j++)
        {

            w2cp[i*w2size + j] = w2[i][j];
        }
    }
    for(int i=0; i<10; i++)
    {
        out2cp[i] = 0;
    }

    hipMalloc((void**)&dev_a1, a1cp_Size);
    hipMalloc((void**)&dev_w2, w2cp_Size);
    hipMalloc((void**)&dev_out2, outSize);

    hipMemcpy(dev_a1, a1cp, a1cp_Size, hipMemcpyHostToDevice);
    hipMemcpy(dev_w2, w2cp, w2cp_Size, hipMemcpyHostToDevice);
    hipMemcpy(dev_out2, out2cp, outSize, hipMemcpyHostToDevice);

    dim3 Dg2(4, 25, 1);
    dim3 Db2(1, 1, 1);
    cout << "두번째 dot" <<endl;
    Dot2 <<<Dg2, Db2>>> (dev_a1, dev_w2, dev_out2, 10, 10);
    hipDeviceSynchronize();
    hipMemcpy(out2cp, dev_out2, outSize, hipMemcpyDeviceToHost);

    out.clear();
    out.resize(1);
    for(int i=0; i<10; i++)
    {
        out[0].push_back((float)out2cp[i]);
        //cout << "out: " << out[0][i] << " " << typeid(out[0][i]).name() << endl;
    }

    cout << out.size() << " " << out[0].size() << " " << outSize/sizeof(float) << endl;

    // printResult(a1cp, w2cp, out2cp);
    // out = dot(a1, w2); //w2:(10,10), out:(1,10)
    sumb(out, b2, out);

    cout << out.size() << " " << out[0].size() << " " << outSize/sizeof(float) << endl;
    //summ(out, b2, out); //b2,out: (1,10)
    hipFree(dev_a1); hipFree(dev_w2); hipFree(dev_out2);
    free(a1cp); free(w2cp); free(out2cp);
    
    return out;  //(1,10)
}

__global__ void SoftMax(float* a, float* out){

    int tid, tx, ty;
    tx = blockDim.x * blockIdx.x + threadIdx.x;
    ty = blockDim.y * blockIdx.y + threadIdx.y;
    int DimX = gridDim.x * blockDim.x;
    tid = DimX * ty + tx;

    float total = 0.0f; float ZVal = 0;

    out[tid] = exp(out[tid]);
    total += out[tid];
    out[tid] /= total;

}


vector<vector<float>> softmax(vector<vector<float>> a){
    float total = 0.0f;
    vector<vector<float>> out;

    for(int i=0; i<a.size(); i++)
    {
        out.push_back(vector<float>());
        for(int j=0;j<a[i].size();j++)
        {
            out[i].push_back(exp(a[i][j]));
            total += out[i].back();
        }
    }

    for(int i=0;i<out.size();i++)
    {
        for(int j=0;j<out[i].size();j++)
        {
            out[i][j] /= total;
        }
    }

    return out;
}

vector<vector<float>> y_train_encoded(vector<float> &y){
    vector<vector<float>> output(y.size(), vector<float>(10, 0));

    for(int i=0;i<output.size();i++)
    {
        output[i][int(y[i])] = 1;
    }
    return output;
}

void backprop(vector<vector<float>> x, vector<vector<float>> err, vector<vector<float>> &w2, vector<vector<float>> &w1_grad,\
             vector<vector<float>> &w2_grad, vector<float> &b1_grad, float &b2_grad, vector<vector<float>> &a1){
    for(int i=0; i<err.size(); i++)
    {
        b2_grad = 0.0f;
        for(int j=0; j<err[i].size(); j++)
        {
            b2_grad += err[i][j];
        }
    }

    vector<vector<float>> a1T = transpose(a1);
    w2_grad = dot(a1T, err);

    vector<vector<float>> w2T = transpose(w2);
    vector<vector<float>> err_to_hidden = dot(err, w2T); //err_to_hidden(1, 10)

    for(int i=0; i<a1.size(); i++)
    {
        for(int j=0; j<a1[i].size(); j++)
        {
            err_to_hidden[i][j] *= a1[i][j] * (1-a1[i][j]);
        }
    }

    vector<vector<float>> xT = transpose(x);
    w1_grad = dot(xT, err_to_hidden);
    
    for(int j=0; j<err_to_hidden[0].size(); j++)
    {
        b1_grad[j] = 0.0f;
        for(int i=0; i<err_to_hidden.size(); i++)
        {
            b1_grad[j] += err_to_hidden[i][j];
        }
    }
}

vector<vector<float>> training(vector<vector<float>> x, vector<vector<float>> y, vector<vector<float>> &w1, \
                    vector<vector<float>> &w2, vector<vector<float>> &w1_grad, vector<vector<float>> &w2_grad, \
                    vector<float> &b1, vector<float> &b2, vector<float> &b1_grad, \
                    float &b2_grad, vector<vector<float>> &a1)
{   
    float* zcp; float* soutcp;
    
    vector<vector<float>> z = forward(x, w1, w2, a1);

    vector<vector<float>> out_softmax;
    int zcp_Size = z.size()*z[0].size()*sizeof(float);

    cout << z.size() << " " << z[0].size() << endl;
    // for(int j=0; j<10; j++)
    // {
    //     cout << z[0][j] << " ";
    // }
    // cout << endl;

    cout << "softmax!" << endl;
    zcp = (float*)malloc(zcp_Size);
    soutcp = (float*)malloc(zcp_Size);
    
    for(int i=0; i<z.size(); i++)
    {
        for(int j=0; j<z[i].size(); j++)
        {
            zcp[i*z[i].size() + j] = z[i][j];
        }
    }

    hipMalloc((void**)&dev_z, zcp_Size);
    hipMalloc((void**)&dev_sout, zcp_Size);

    for(int i=0; i<10; i++)
    {
        soutcp[i] = 0;
    }
    hipMemcpy(dev_z, zcp, zcp_Size, hipMemcpyHostToDevice);
    hipMemcpy(dev_sout, soutcp, zcp_Size, hipMemcpyHostToDevice);

    dim3 Dg(1, 10, 1);
    dim3 Db(10, 10, 1);

    SoftMax <<<Dg, Db>>> (dev_z, dev_out);
    hipDeviceSynchronize();
    hipMemcpy(outcp, dev_out, zcp_Size, hipMemcpyDeviceToHost);

    out_softmax.resize(1);
    for(int i=0; i<10; i++)
    {
        out_softmax[0].push_back((float)soutcp[i]);
        //cout << "out: " << out[0][i] << " " << typeid(out[0][i]).name() << endl;
    }

    printResult(dev_z, dev_sout, dev_sout);

    hipFree(dev_z); hipFree(dev_sout);
    free(zcp); free(soutcp);

    //vector<vector<float>> a = softmax(z);
    vector<vector<float>> err(y.size());

    for(int i=0; i<y.size(); i++)
    {
        for(int j=0; j<y[i].size(); j++)
        {
            err[i].push_back(-(y[i][j]-out_softmax[i][j]));
        }
    }

    cout << "backprop!" << endl;
    backprop(x, err, w2, w1_grad, w2_grad, b1_grad, b2_grad, a1);

    cout << "tooooo busy" << endl;
    vector<vector<int>> sign1(w1.size(), vector<int>(w1[0].size(), 0));
    vector<vector<int>> sign2(w2.size(), vector<int>(w2[0].size(), 0));
    for(int i=0; i<w1.size(); i++)
    {
        for(int j=0; j<w1[i].size(); j++)
        {
            sign1[i][j] = (fabs(w1[i][j]) < 1e-9) ? 0 : round(w1[i][j]/(fabs(w1[i][j])));
        }
    }
    for(int i=0; i<w2.size(); i++)
    {
        for(int j=0; j<w2[i].size(); j++)
        {
            sign2[i][j] = (fabs(w2[i][j]) < 1e-9) ? 0 : round(w2[i][j]/(fabs(w2[i][j])));
        }
    }
    
    float lr = 0.1;

    for(int i=0; i<w1_grad.size(); i++)
    {
        for(int j=0; j<w1_grad[i].size(); j++)
        {
            w1_grad[i][j] += (l1*sign1[i][j] + l2*w1[i][j]);
        }
    }
    for(int i=0; i<w2_grad.size(); i++)
    {
        for(int j=0; j<w2_grad[i].size(); j++)
        {
            w2_grad[i][j] += (l1*sign2[i][j] + l2*w2[i][j]);
        }
    }

    for(int i=0;i<w1.size();i++)
    {
        for(int j=0;j<w1[i].size();j++)
        {
            w1[i][j] -= lr * w1_grad[i][j];
        }
    }
    for(int i=0;i<w2.size();i++)
    {
        for(int j=0;j<w2[i].size();j++)
        {
            w2[i][j] -= lr * w2_grad[i][j];
        }
    }
    for(int i=0;i<b1.size();i++)
    {
        b1[i] -= lr * b1_grad[i];
    }
    for(int i=0;i<b1.size();i++)
    {
        b2[i] -= lr * b2_grad;
    }

    return out_softmax;
}

float reg_loss(vector<vector<float>> w1, vector<vector<float>> w2)
{
    float tmp1 = 0.0f;

    for(int i=0; i<w1[0].size(); i++) //~10
    {
        for(int j=0; j<w1.size(); j++) //~784
        {
            if(w1[j][i] < 0) w1[j][i] = -w1[j][i];
            
            tmp1 += l1*(w1[j][i]) + l2/2*pow(w1[j][i],2);
            //tmp1 += l1*(w1[i][j]) + w2[i][j] + l2/2*pow(w1[i][j],2) + pow(w2[i][j],2);
        }

        for(int j=0; j<w2[i].size(); j++) //~10
        {
            if(w2[i][j] < 0) w2[i][j] = -w2[i][j];
            tmp1 += w2[i][j] + pow(w2[i][j],2);
        }
    }

    return tmp1;
}

void update_val_loss(vector<vector<float>> x_val, vector<vector<float>> y_val, vector<vector<float>> &w1, vector<vector<float>> &w2, \
                    vector<float> &b1, vector<float> &b2, vector<vector<float>> &a1)
{
    vector<vector<float>> z = forward(x_val, w1, w2, a1);
    // for(int j=0; j<z.size(); j++)
    // {
    //     for(int k=0; k<z[0].size(); k++)
    //     cout << z[j][k] << " ";
    // }
    // cout << endl;
    vector<vector<float>> a = softmax(z);
    float val_loss = 0.0f;
    for(int ii=0;ii<a.size();ii++)
    {
        for(int jj=0;jj<a[ii].size();jj++)
        {
            val_loss += (-y_val[ii][jj] * log(a[ii][jj]));
        }
    }
    val_losses.push_back((val_loss+reg_loss(w1, w2))/y_val.size());
}


void fit(vector<vector<float>> &x_val, vector<vector<float>> &y_val, vector<vector<float>> &x, vector<vector<float>> &y, int epochs){
    vector<vector<float>> trained_a(10);
    vector<float> loss(trained_a.size());
    for(int i=0; i<epochs; i++)
    {
        float loss = 0.0f;
        cout << ".";

        vector<vector<float>> smallx(1, vector<float>(784, 0.0)), smally(1, vector<float>(10, 0));
        // cout << "x size is " << x.size() << endl;
        int x_sizee = x.size();
        for(int j=0; j<x_sizee; j++)
        {
            smallx[0].assign(x[j].begin(), x[j].end());
            smally[0].assign(y[j].begin(), y[j].end());
            // copy(x_val[i].begin(), x_val[i].end(), smallx[0].begin());
            // copy(y_val[i].begin(), y_val[i].end(), smally[0].begin());

            trained_a = training(smallx, smally, w1, w2, w1_grad, w2_grad, b1, b2, b1_grad, b2_grad, a1_train);

            // cout << "after training" << endl;
            float loss = 0.0f;
            for(int ii=0;ii<trained_a.size();ii++)
            {
                // cout << "first for" << endl;
                for(int jj=0;jj<trained_a[ii].size();jj++)
                {
                    // cout << "second for" << endl;
                    loss += (-y_val[ii][jj] * log(trained_a[ii][jj]));
                }
            }
            // cout << "end for" << endl;
        }
        cout << "train finish" <<endl;
        losses.push_back((loss+reg_loss(w1, w2))/smally.size());

        for(int j=0;j<x_val.size();j++)
        {
            smallx[0].assign(x_val[j].begin(), x_val[j].end());
            smally[0].assign(y_val[j].begin(), y_val[j].end());
            update_val_loss(smallx, smally, w1, w2, b1, b2, a1_train);
        }
        // update_val_loss(x_val, y_val, w1, w2, b1, b2, a1_val);
        
    }
    cout << endl;
}

// n_in=784, n_out=10
void kaiming_init(vector<vector<float>> &w, int n_in){
    float std = sqrt(2/(float) n_in);
    
    random_device rd;
    mt19937 gen(rd()); 
    normal_distribution<float> dist(0.0f, std); 

    for (int i=0; i<n_in; i++){
        for(int j=0; j<10; j++)
        {
            //cout << "(" << i << ", " << j << ") " << dist(gen) << "\t";
            w[i].push_back(dist(gen));
        }
    }
}

vector<int> predict(vector<vector<float>> x, vector<vector<float>> &w1, vector<vector<float>> &w2){
    vector<int> result;
    vector<vector<float>> smallx(1);

    for(int i=0; i<x.size(); i++)
    {
        smallx[0].assign(x[i].begin(), x[i].end());
        //cout << smallx[0][347] << ' ';
        vector<vector<float>> z = forward(smallx, w1, w2, a1_train);

        float tmp = -1e9;
        int idx = 0;
        for(int j=0; j<z[0].size(); j++)
        {
            if(tmp < z[0][j])
            {
                tmp = z[0][j];
                idx = j;
            }
        }
        result.push_back(idx);

        // for(int j=0; j<z[0].size(); j++)
        // {
        //     cout << z[0][j] << " ";
        // }
        // cout << endl;
    }

    // for(int i=0; i<result.size(); i++)
    // {
    //     cout << result[i] << endl;
    // }
    cout << "finish predict" << endl;
    return result;
}

float score(vector<int> result, vector<vector<float>> y)
{
    //vector<vector<float>> smally(1, vector<float>(784, 0.0));
    vector<int> resulty;

    for(int i=0; i<y.size(); i++)
    {
        float tmp = 0.0f;
        int idx = 0;    
        for(int j=0;j<y[i].size();j++)
        {
            if(tmp < y[i][j])
            {
                tmp = y[i][j];
                idx = j;
            }
        }
        resulty.push_back(idx);
        //smally[0].assign(y[i].begin(), y[i].end());
        // float tmp = 0.0f;
        // int idx = 0;
        // for(int j=0; j<smally[0].size(); j++)
        // {
        //     if(tmp < smally[0][j])
        //     {
        //         tmp = smally[0][j];
        //         idx = j;
        //     }
        // }
    }
    // for(int i=0;i<result.size();i++)
    //     cout << result[i] << ' ';

    int cnt = 0;
    for(int i=0; i<result.size(); i++)
    {
        if(result[i] == resulty[i])
        {
            cnt++;
        }
    }

    // for(int i=0; i<100; i++)
    // {
    //     cout << result[i] << " vs " << resulty[i] << endl;
    // }

    float score = (float) cnt/result.size();
    return score;
}


int main()
{
    chrono::steady_clock::time_point begin, end;

    // 데이터 읽기 !
    begin = chrono::steady_clock::now();
    ifstream readFile;             //읽을 목적의 파일 선언
    readFile.open("train.csv");    //파일 열기

    int idx=0;

    if(readFile.is_open())    //파일이 열렸는지 확인
    {
        // 맨 윗줄 제거
        string row;
        getline(readFile, row);

        cout << "start load train data ...\n";

        while(!readFile.eof())    //파일 끝까지 읽었는지 확인
        {
            getline(readFile, row);
            istringstream ss(row);

            string num;
            
            while(getline(ss, num, ','))
            {
                //int a = atoi(num.c_str());
                float a = stof(num);
                load_data[idx].push_back(a);
            }
            
            idx++;
        }
        cout << "train_row: " << load_data.size() << ", train_column: " << load_data[0].size() << "\n";
        cout << "finish load train_data !\n";
        readFile.close();
    }
    else
        cout << "Can not open file\n";

    // X_train, y_train 만들기
    y_train.clear();

    for(int i=0; i<26880; i++)
    {
        for(int j=1; j<load_data[i].size(); j++)
        {
            X_train[i].push_back(load_data[i][j]);
        }
        float a = load_data[i][0];
        y_train.push_back(a);
    }


    //val 파일 읽기
    load_data.clear();
    load_data.resize(8400);

    readFile.open("val.csv");    //파일 열기

    idx=0;

    if(readFile.is_open())    //파일이 열렸는지 확인
    {
        // 맨 윗줄 제거
        string row;
        getline(readFile, row);

        cout << "start load test data ...\n";

        while(!readFile.eof())    //파일 끝까지 읽었는지 확인
        {
            getline(readFile, row);
            istringstream ss(row);

            string num;
            
            while(getline(ss, num, ','))
            {
                //int a = atoi(num.c_str());
                float a = stof(num);
                load_data[idx].push_back(a);
            }
            
            idx++;
        }
        cout << "val_row: " << load_data.size() << ", val_column: " << load_data[0].size() << "\n";
        cout << "finish load val_data !\n";
        readFile.close();
    }
    else
        cout << "Can not open file\n";

    
    x_val.resize(8400);
    // X_train, y_train 만들기
    //y_train.clear();
    //cout << "x_val size: " << x_val.size() << " " << x_val[0].size() << " " << y_val.size() << " " << y_val[0].size() << " " << load_data[0].size();
    for(int i=0; i<8400; i++)
    {
        for(int j=1; j<load_data[i].size(); j++)
           x_val[i].push_back(load_data[i][j]);

        float a = load_data[i][0]; //꼭 int로 해야하나?
        y_val.push_back(a);
    }

    // test 데이터 불러오기
    load_data.clear();
    load_data.resize(6720);


    readFile.open("test.csv");    //파일 열기

    idx=0;

    if(readFile.is_open())    //파일이 열렸는지 확인
    {
        // 맨 윗줄 제거
        string row;
        getline(readFile, row);

        cout << "start load train data ...\n";

        while(!readFile.eof())    //파일 끝까지 읽었는지 확인
        {
            getline(readFile, row);
            istringstream ss(row);

            string num;
            
            while(getline(ss, num, ','))
            {
                //int a = atoi(num.c_str());
                float a = stof(num);
                load_data[idx].push_back(a);
            }
            
            idx++;
        }
        cout << "test_row: " << load_data.size() << ", test_column: " << load_data[0].size() << "\n";
        cout << "finish load test_data !\n";
        readFile.close();
    }
    else
        cout << "Can not open file\n";


    for(int i=0; i<6720; i++)
    {
        for(int j=1; j<load_data[i].size(); j++)
        {
            x_test[i].push_back(load_data[i][j]);
        }

        float a = load_data[i][0];
        y_test.push_back(a);
    }
    //cout << "X_train size: " << X_train.size() << " " << X_train[0].size() << " " << y_train.size() << " " << load_data[0].size();


    end = chrono::steady_clock::now();
    cout << "Data reading time: " << (chrono::duration_cast<chrono::microseconds>(end-begin).count())/1000000.0f << endl;


    
    kaiming_init(w1, w1.size());
    kaiming_init(w2, w2.size());

    
    vector<vector<float>> y_train_enc = y_train_encoded(y_train);
    vector<vector<float>> y_val_enc = y_train_encoded(y_val);
    vector<vector<float>> y_test_enc = y_train_encoded(y_test);

    // hipMalloc((void**)&dev_xx, xSize);
    // hipMalloc((void**)&dev_w1, w1Size);
    // hipMalloc((void**)&dev_out, outSize);

    fit(x_val, y_val_enc, X_train, y_train_enc, 1);

    // hipFree(dev_xx); hipFree(dev_w1); hipFree(dev_out);
    // free(xcp); free(w1cp); free(outcp);
    cout << "training end ~ " << endl;

    vector<int> result;
    result = predict(x_test, w1, w2);
    cout << "predict finish" << endl;

    // for(int i=0; i<result.size(); i++)
    // {
    //     cout << result[i] << endl;
    // }
    float plz = score(result, y_test_enc);
    cout << "score : " << plz << endl;
    cout << "the end ~ " << endl;

    return 0;
}