#include "hip/hip_runtime.h"
%%writefile test2.cu
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <sstream>
#include <chrono>
#include <cmath>
#include <cstdlib>
#include <random>
#include <typeinfo>

using namespace std;

vector<vector<float>> load_data(26880);
vector<vector<float>> X_train(26880);
vector<vector<float>> x_val(8400);
vector<vector<float>> x_test(6720);
vector<float> y_train;
vector<float> y_val;
vector<float> y_test;

vector<float> losses, val_losses;
vector<vector<float>> w1(784), w2(10);
vector<float> b1(10), b2(10);
vector<vector<float>> output(10), a1_train(1, vector<float>(10, 0)), a1_test(1, vector<float>(10, 0)), a1_val(1, vector<float>(10, 0));
vector<float> b1_grad(10);
float b2_grad;
vector<vector<float>> w1_grad(10), w2_grad(10); // w2_grad (10, 10)

float* dev_w1; float* dev_out;
float* dev_xx;
int xSize = 1*784 * sizeof(float);
int w1Size = 784*10 * sizeof(float);
int outSize = 1*10 * sizeof(float);

float* xcp; float* w1cp; float* outcp;


float l1 = 0, l2 = 0;



vector<vector<float>> transpose(vector<vector<float>>& a)
{
    vector<vector<float>> ret(a[0].size(), vector<float>(a.size()));
    for(int i=0;i<a.size();i++)
    {
        for(int j=0;j<a[i].size();j++)
        {
            ret[j][i] = a[i][j];
        }
    }
    return ret;
}
vector<vector<float>> dot(vector<vector<float>> &a, vector<vector<float>>& b)
{
    vector<vector<float>> out(a.size(), vector<float>(b[0].size(), 0.0f));

    for(int i=0;i<a.size();i++)
    {
        for(int j=0;j<b[0].size();j++)
        {
            for(int k=0;k<b.size();k++)
            {
                out[i][j] += a[i][k] * b[k][j];
            }
        }
    }
    return out;
}

__global__ void Dot2(float* a, float* b, float* out, int a_width, int b_width)
{
    int tid, tx, ty;
    tx = blockDim.x * blockIdx.x + threadIdx.x;
    ty = blockDim.y * blockIdx.y + threadIdx.y;
    int DimX = gridDim.x * blockDim.x;
    tid = DimX * ty + tx;

    float Value = 0; float AVal = 0; float BVal = 0;

    for (int k = 0; k < a_width; k++)
    {
        AVal = a[ty * a_width + k];
        BVal = b[k * b_width + tx];
        Value += AVal * BVal;
        // printf("\n tid: %d, k: %d, Value: %f, MVal: %f, NVal: %f ", tid, k, Value, AVal, BVal);
    }

    out[tid] = Value;
}


void summ(vector<vector<float>> a, vector<vector<float>> b, vector<vector<float>>& out){   
    for(int i=0; i<a.size(); i++)
    {
        for(int j=0; j<a[i].size(); j++)
        {
            out[i][j] = a[i][j] + b[i][j];
        }
    }
}


void sumb(vector<vector<float>> a, vector<float> b, vector<vector<float>>& out)
{
    for(int i=0;i<a.size();i++)
    {
        for(int j=0;j<a[i].size();j++)
        {
            out[i][j] = a[i][j] + b[j];
        }
    }
}


vector<vector<float>> sigmoid(vector<vector<float>> a){
    vector<vector<float>> out(a.size(), vector<float>(a[0].size(), 0));
    for(int i=0; i<a.size(); i++)
    {
        for(int j=0; j<a[i].size(); j++)
        {
            float tmp = a[i][j] / 4000;
            out[i][j] = 1.0/((float)1.0+exp(-tmp));
        }
    }
    return out;
}

void printResult(float* M, float* N, float* P)
{
    int row = 0; int col = 0;
    for (row = 0; row < 1; row++)
    {
        for (col = 0; col < 10; col++)
        {
            int Destindex = row * 1 + col;
            printf( "%f (= P[%d][%d]) = %f (= M[%d][%d]) + %f (= N[%d][%d]) \n", \
            P[Destindex], row, col, M[Destindex], row, col, N[Destindex], row, col );
        }
    }
}


vector<vector<float>> forward(vector<vector<float>> &x, vector<vector<float>> &w1, vector<vector<float>> &w2, vector<vector<float>> &a1){
    cout << "forward!" << endl;
    vector<vector<float>> out;

    xcp = (float*)malloc(xSize); //(1, 784)
    w1cp = (float*)malloc(w1Size); //(784, 10)
    outcp = (float*)malloc(outSize); //(1, 10)

    cout << "malloc!" << endl;

    int xxsize = x.size();
    int wwsize = w1.size();

    cout << "111!" << endl;
    for(int i=0; i<x.size(); i++)
    {
        for(int j=0; j<x[i].size(); j++)
        {
            xcp[i*xxsize + j] = x[i][j];
        }
    }
    cout << "222!" << endl;
    for(int i=0; i<w1.size(); i++)
    {
        for(int j=0; j<w1[i].size(); j++)
        {

            w1cp[i*wwsize + j] = w1[i][j];
        }
    }
    cout << "333!" << endl;

    for(int i=0; i<10; i++)
    {
        outcp[i] = 0;
    }
    cout << "444!" << endl;


    cout << "cudamalloc!" << endl;
    hipMemcpy(dev_xx, xcp, xSize, hipMemcpyHostToDevice);
    hipMemcpy(dev_w1, w1cp, w1Size, hipMemcpyHostToDevice);
    hipMemcpy(dev_out, outcp, outSize, hipMemcpyHostToDevice);
    cout << "hipMemcpy!" << endl;

    dim3 Dg(1, 10, 1);
    dim3 Db(128, 1, 1);

    cout << "before Dot2!" << endl;
    Dot2 <<<Dg, Db>>> (dev_xx, dev_w1, dev_out, 784, 10);
    hipDeviceSynchronize();
    cout << "after Dot2!" << endl;
    hipMemcpy(outcp, dev_out, outSize, hipMemcpyDeviceToHost);

    out.resize(1);
    for(int i=0; i<10; i++)
    {
        out[0].push_back((float)outcp[i]);
        //cout << "out: " << out[0][i] << " " << typeid(out[0][i]).name() << endl;
    }

    printResult(xcp, w1cp, outcp);

    hipFree(dev_xx); hipFree(dev_w1); hipFree(dev_out);
    free(xcp); free(w1cp); free(outcp);


    out = dot(x, w1);
    sumb(out, b1, out); //(1, 10)
    //cout << out[0][0] << endl;
    a1 = sigmoid(out); //(1,10)
    out = dot(a1, w2); //w2:(10,10), out:(1,10)
    sumb(out, b2, out);
    //summ(out, b2, out); //b2,out: (1,10)
    
    return out;  //(1,10)
}


vector<vector<float>> softmax(vector<vector<float>> a){
    float total = 0.0f;
    vector<vector<float>> out;

    for(int i=0; i<a.size(); i++)
    {
        out.push_back(vector<float>());
        for(int j=0;j<a[i].size();j++)
        {
            out[i].push_back(exp(a[i][j]));
            total += out[i].back();
        }
    }

    for(int i=0;i<out.size();i++)
    {
        for(int j=0;j<out[i].size();j++)
        {
            out[i][j] /= total;
        }
    }

    return out;
}

vector<vector<float>> y_train_encoded(vector<float> &y){
    vector<vector<float>> output(y.size(), vector<float>(10, 0));

    for(int i=0;i<output.size();i++)
    {
        output[i][int(y[i])] = 1;
    }
    return output;
}

void backprop(vector<vector<float>> x, vector<vector<float>> err, vector<vector<float>> &w2, vector<vector<float>> &w1_grad,\
             vector<vector<float>> &w2_grad, vector<float> &b1_grad, float &b2_grad, vector<vector<float>> &a1){
    for(int i=0; i<err.size(); i++)
    {
        b2_grad = 0.0f;
        for(int j=0; j<err[i].size(); j++)
        {
            b2_grad += err[i][j];
        }
    }

    vector<vector<float>> a1T = transpose(a1);
    w2_grad = dot(a1T, err);

    vector<vector<float>> w2T = transpose(w2);
    vector<vector<float>> err_to_hidden = dot(err, w2T); //err_to_hidden(1, 10)

    for(int i=0; i<a1.size(); i++)
    {
        for(int j=0; j<a1[i].size(); j++)
        {
            err_to_hidden[i][j] *= a1[i][j] * (1-a1[i][j]);
        }
    }

    vector<vector<float>> xT = transpose(x);
    w1_grad = dot(xT, err_to_hidden);
    
    for(int j=0; j<err_to_hidden[0].size(); j++)
    {
        b1_grad[j] = 0.0f;
        for(int i=0; i<err_to_hidden.size(); i++)
        {
            b1_grad[j] += err_to_hidden[i][j];
        }
    }
}

vector<vector<float>> training(vector<vector<float>> x, vector<vector<float>> y, vector<vector<float>> &w1, \
                    vector<vector<float>> &w2, vector<vector<float>> &w1_grad, vector<vector<float>> &w2_grad, \
                    vector<float> &b1, vector<float> &b2, vector<float> &b1_grad, \
                    float &b2_grad, vector<vector<float>> &a1)
{
    vector<vector<float>> z = forward(x, w1, w2, a1);
    // for(int j=0; j<10; j++)
    // {
    //     cout << z[0][j] << " ";
    // }
    // cout << endl;

    cout << "softmax!" << endl;
    vector<vector<float>> a = softmax(z);
    vector<vector<float>> err(y.size());

    for(int i=0; i<y.size(); i++)
    {
        for(int j=0; j<y[i].size(); j++)
        {
            err[i].push_back(-(y[i][j]-a[i][j]));
        }
    }

    cout << "backprop!" << endl;
    backprop(x, err, w2, w1_grad, w2_grad, b1_grad, b2_grad, a1);

    cout << "tooooo busy" << endl;
    vector<vector<int>> sign1(w1.size(), vector<int>(w1[0].size(), 0));
    vector<vector<int>> sign2(w2.size(), vector<int>(w2[0].size(), 0));
    for(int i=0; i<w1.size(); i++)
    {
        for(int j=0; j<w1[i].size(); j++)
        {
            sign1[i][j] = (fabs(w1[i][j]) < 1e-9) ? 0 : round(w1[i][j]/(fabs(w1[i][j])));
        }
    }
    for(int i=0; i<w2.size(); i++)
    {
        for(int j=0; j<w2[i].size(); j++)
        {
            sign2[i][j] = (fabs(w2[i][j]) < 1e-9) ? 0 : round(w2[i][j]/(fabs(w2[i][j])));
        }
    }
    
    float lr = 0.1;

    for(int i=0; i<w1_grad.size(); i++)
    {
        for(int j=0; j<w1_grad[i].size(); j++)
        {
            w1_grad[i][j] += (l1*sign1[i][j] + l2*w1[i][j]);
        }
    }
    for(int i=0; i<w2_grad.size(); i++)
    {
        for(int j=0; j<w2_grad[i].size(); j++)
        {
            w2_grad[i][j] += (l1*sign2[i][j] + l2*w2[i][j]);
        }
    }

    for(int i=0;i<w1.size();i++)
    {
        for(int j=0;j<w1[i].size();j++)
        {
            w1[i][j] -= lr * w1_grad[i][j];
        }
    }
    for(int i=0;i<w2.size();i++)
    {
        for(int j=0;j<w2[i].size();j++)
        {
            w2[i][j] -= lr * w2_grad[i][j];
        }
    }
    for(int i=0;i<b1.size();i++)
    {
        b1[i] -= lr * b1_grad[i];
    }
    for(int i=0;i<b1.size();i++)
    {
        b2[i] -= lr * b2_grad;
    }

    return a;
}

float reg_loss(vector<vector<float>> w1, vector<vector<float>> w2)
{
    float tmp1 = 0.0f;

    for(int i=0; i<w1[0].size(); i++) //~10
    {
        for(int j=0; j<w1.size(); j++) //~784
        {
            if(w1[j][i] < 0) w1[j][i] = -w1[j][i];
            
            tmp1 += l1*(w1[j][i]) + l2/2*pow(w1[j][i],2);
            //tmp1 += l1*(w1[i][j]) + w2[i][j] + l2/2*pow(w1[i][j],2) + pow(w2[i][j],2);
        }

        for(int j=0; j<w2[i].size(); j++) //~10
        {
            if(w2[i][j] < 0) w2[i][j] = -w2[i][j];
            tmp1 += w2[i][j] + pow(w2[i][j],2);
        }
    }

    return tmp1;
}

void update_val_loss(vector<vector<float>> x_val, vector<vector<float>> y_val, vector<vector<float>> &w1, vector<vector<float>> &w2, \
                    vector<float> &b1, vector<float> &b2, vector<vector<float>> &a1)
{
    vector<vector<float>> z = forward(x_val, w1, w2, a1);
    // for(int j=0; j<z.size(); j++)
    // {
    //     for(int k=0; k<z[0].size(); k++)
    //     cout << z[j][k] << " ";
    // }
    // cout << endl;
    vector<vector<float>> a = softmax(z);
    float val_loss = 0.0f;
    for(int ii=0;ii<a.size();ii++)
    {
        for(int jj=0;jj<a[ii].size();jj++)
        {
            val_loss += (-y_val[ii][jj] * log(a[ii][jj]));
        }
    }
    val_losses.push_back((val_loss+reg_loss(w1, w2))/y_val.size());


}


void fit(vector<vector<float>> &x_val, vector<vector<float>> &y_val, vector<vector<float>> &x, vector<vector<float>> &y, int epochs){
    vector<vector<float>> trained_a(10);
    vector<float> loss(trained_a.size());
    for(int i=0; i<epochs; i++)
    {
        float loss = 0.0f;
        cout << ".";

        vector<vector<float>> smallx(1, vector<float>(784, 0.0)), smally(1, vector<float>(10, 0));
        cout << "x size is " << x.size() << endl;
        int x_sizee = x.size();
        for(int j=0; j<x_sizee; j++)
        {
            smallx[0].assign(x[j].begin(), x[j].end());
            smally[0].assign(y[j].begin(), y[j].end());
            // copy(x_val[i].begin(), x_val[i].end(), smallx[0].begin());
            // copy(y_val[i].begin(), y_val[i].end(), smally[0].begin());

            trained_a = training(smallx, smally, w1, w2, w1_grad, w2_grad, b1, b2, b1_grad, b2_grad, a1_train);

            cout << "after training" << endl;
            float loss = 0.0f;
            for(int ii=0;ii<trained_a.size();ii++)
            {
                cout << "first for" << endl;
                for(int jj=0;jj<trained_a[ii].size();jj++)
                {
                    cout << "second for" << endl;
                    loss += (-y_val[ii][jj] * log(trained_a[ii][jj]));
                }
            }
            cout << "end for" << endl;
        }
        
        losses.push_back((loss+reg_loss(w1, w2))/smally.size());

        for(int j=0;j<x_val.size();j++)
        {
            smallx[0].assign(x_val[j].begin(), x_val[j].end());
            smally[0].assign(y_val[j].begin(), y_val[j].end());
            update_val_loss(smallx, smally, w1, w2, b1, b2, a1_train);
        }
        // update_val_loss(x_val, y_val, w1, w2, b1, b2, a1_val);
    }
    cout << endl;
}

// n_in=784, n_out=10
void kaiming_init(vector<vector<float>> &w, int n_in){
    float std = sqrt(2/(float) n_in);
    
    random_device rd;
    mt19937 gen(rd()); 
    normal_distribution<float> dist(0.0f, std); 

    for (int i=0; i<n_in; i++){
        for(int j=0; j<10; j++)
        {
            //cout << "(" << i << ", " << j << ") " << dist(gen) << "\t";
            w[i].push_back(dist(gen));
        }
    }
}

vector<int> predict(vector<vector<float>> x, vector<vector<float>> &w1, vector<vector<float>> &w2){
    vector<int> result;
    vector<vector<float>> smallx(1);

    for(int i=0; i<x.size(); i++)
    {
        smallx[0].assign(x[i].begin(), x[i].end());
        //cout << smallx[0][347] << ' ';
        vector<vector<float>> z = forward(smallx, w1, w2, a1_train);

        float tmp = -1e9;
        int idx = 0;
        for(int j=0; j<z[0].size(); j++)
        {
            if(tmp < z[0][j])
            {
                tmp = z[0][j];
                idx = j;
            }
        }
        result.push_back(idx);

        // for(int j=0; j<z[0].size(); j++)
        // {
        //     cout << z[0][j] << " ";
        // }
        // cout << endl;
    }

    // for(int i=0; i<result.size(); i++)
    // {
    //     cout << result[i] << endl;
    // }
    cout << "finish predict" << endl;
    return result;
}

float score(vector<int> result, vector<vector<float>> y)
{
    //vector<vector<float>> smally(1, vector<float>(784, 0.0));
    vector<int> resulty;

    for(int i=0; i<y.size(); i++)
    {
        float tmp = 0.0f;
        int idx = 0;    
        for(int j=0;j<y[i].size();j++)
        {
            if(tmp < y[i][j])
            {
                tmp = y[i][j];
                idx = j;
            }
        }
        resulty.push_back(idx);
        //smally[0].assign(y[i].begin(), y[i].end());
        // float tmp = 0.0f;
        // int idx = 0;
        // for(int j=0; j<smally[0].size(); j++)
        // {
        //     if(tmp < smally[0][j])
        //     {
        //         tmp = smally[0][j];
        //         idx = j;
        //     }
        // }
    }
    // for(int i=0;i<result.size();i++)
    //     cout << result[i] << ' ';

    int cnt = 0;
    for(int i=0; i<result.size(); i++)
    {
        if(result[i] == resulty[i])
        {
            cnt++;
        }
    }

    // for(int i=0; i<100; i++)
    // {
    //     cout << result[i] << " vs " << resulty[i] << endl;
    // }

    float score = (float) cnt/result.size();
    return score;
}


int main()
{
    chrono::steady_clock::time_point begin, end;

    // 데이터 읽기 !
    begin = chrono::steady_clock::now();
    ifstream readFile;             //읽을 목적의 파일 선언
    readFile.open("/content/drive/MyDrive/NN/train.csv");    //파일 열기

    int idx=0;

    if(readFile.is_open())    //파일이 열렸는지 확인
    {
        // 맨 윗줄 제거
        string row;
        getline(readFile, row);

        cout << "start load train data ...\n";

        while(!readFile.eof())    //파일 끝까지 읽었는지 확인
        {
            getline(readFile, row);
            istringstream ss(row);

            string num;
            
            while(getline(ss, num, ','))
            {
                //int a = atoi(num.c_str());
                float a = stof(num);
                load_data[idx].push_back(a);
            }
            
            idx++;
        }
        cout << "train_row: " << load_data.size() << ", train_column: " << load_data[0].size() << "\n";
        cout << "finish load train_data !\n";
        readFile.close();
    }
    else
        cout << "Can not open file\n";

    // X_train, y_train 만들기
    y_train.clear();

    for(int i=0; i<26880; i++)
    {
        for(int j=1; j<load_data[i].size(); j++)
        {
            X_train[i].push_back(load_data[i][j]);
        }
        float a = load_data[i][0];
        y_train.push_back(a);
    }


    //val 파일 읽기
    load_data.clear();
    load_data.resize(8400);

    readFile.open("/content/drive/MyDrive/NN/val.csv");    //파일 열기

    idx=0;

    if(readFile.is_open())    //파일이 열렸는지 확인
    {
        // 맨 윗줄 제거
        string row;
        getline(readFile, row);

        cout << "start load test data ...\n";

        while(!readFile.eof())    //파일 끝까지 읽었는지 확인
        {
            getline(readFile, row);
            istringstream ss(row);

            string num;
            
            while(getline(ss, num, ','))
            {
                //int a = atoi(num.c_str());
                float a = stof(num);
                load_data[idx].push_back(a);
            }
            
            idx++;
        }
        cout << "val_row: " << load_data.size() << ", val_column: " << load_data[0].size() << "\n";
        cout << "finish load val_data !\n";
        readFile.close();
    }
    else
        cout << "Can not open file\n";

    
    x_val.resize(8400);
    // X_train, y_train 만들기
    //y_train.clear();
    //cout << "x_val size: " << x_val.size() << " " << x_val[0].size() << " " << y_val.size() << " " << y_val[0].size() << " " << load_data[0].size();
    for(int i=0; i<8400; i++)
    {
        for(int j=1; j<load_data[i].size(); j++)
           x_val[i].push_back(load_data[i][j]);

        float a = load_data[i][0]; //꼭 int로 해야하나?
        y_val.push_back(a);
    }

    // test 데이터 불러오기
    load_data.clear();
    load_data.resize(6720);


    readFile.open("/content/drive/MyDrive/NN/test.csv");    //파일 열기

    idx=0;

    if(readFile.is_open())    //파일이 열렸는지 확인
    {
        // 맨 윗줄 제거
        string row;
        getline(readFile, row);

        cout << "start load train data ...\n";

        while(!readFile.eof())    //파일 끝까지 읽었는지 확인
        {
            getline(readFile, row);
            istringstream ss(row);

            string num;
            
            while(getline(ss, num, ','))
            {
                //int a = atoi(num.c_str());
                float a = stof(num);
                load_data[idx].push_back(a);
            }
            
            idx++;
        }
        cout << "test_row: " << load_data.size() << ", test_column: " << load_data[0].size() << "\n";
        cout << "finish load test_data !\n";
        readFile.close();
    }
    else
        cout << "Can not open file\n";


    for(int i=0; i<6720; i++)
    {
        for(int j=1; j<load_data[i].size(); j++)
        {
            x_test[i].push_back(load_data[i][j]);
        }

        float a = load_data[i][0];
        y_test.push_back(a);
    }
    //cout << "X_train size: " << X_train.size() << " " << X_train[0].size() << " " << y_train.size() << " " << load_data[0].size();


    end = chrono::steady_clock::now();
    cout << "Data reading time: " << (chrono::duration_cast<chrono::microseconds>(end-begin).count())/1000000.0f << endl;


    
    kaiming_init(w1, w1.size());
    kaiming_init(w2, w2.size());

    
    vector<vector<float>> y_train_enc = y_train_encoded(y_train);
    vector<vector<float>> y_val_enc = y_train_encoded(y_val);
    vector<vector<float>> y_test_enc = y_train_encoded(y_test);

    hipMalloc((void**)&dev_xx, xSize);
    hipMalloc((void**)&dev_w1, w1Size);
    hipMalloc((void**)&dev_out, outSize);

    fit(x_val, y_val_enc, X_train, y_train_enc, 1);

    hipFree(dev_xx); hipFree(dev_w1); hipFree(dev_out);
    free(xcp); free(w1cp); free(outcp);
    cout << "training end ~ " << endl;


    return 0;
}
